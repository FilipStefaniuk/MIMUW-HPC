#include "hip/hip_runtime.h"
#include <sstream>
#include <iomanip>
#include <cstdlib>
#include <random>
#include "matrix.hh"
#include "matmul.cuh"



Matrix::Matrix(int rows, int cols) : rows(rows), cols(cols) {
    hipMalloc((void**)&(this->buff), BLOCK_ROUND_UP(rows) * BLOCK_ROUND_UP(cols) * sizeof(float));
}

// Matrix::Matrix(unsigned rows, unsigned cols, float *buff) : rows(rows), cols(cols) {
//     hipMalloc((void**)&(this->buff), rows * cols * sizeof(float));
//     hipMemcpy(this->buff, buff, rows * cols * sizeof(float), hipMemcpyHostToDevice);
// }

Matrix::~Matrix() {
    hipFree(this->buff);
}


// __global__ void matMulCUDA(float *A, float *B, float *C, int M, int N, int K) {

//     float Cvalue = 0;

//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {

//         for (int i = 0; i < K; ++i)
//             Cvalue += A[y * K + i] * B[i * N + x];

//         C[y * N + x] = Cvalue;
//     }
// }

void Matrix::matMul(Matrix const &A, Matrix const &B, Matrix &C) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matMulCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(B.cols), BLOCK_ROUND_UP(B.rows));

}

void Matrix::matMulT0(Matrix const &A, Matrix const &B, Matrix &C) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.cols) / dimBlock.y);

    matMulT0CUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, BLOCK_ROUND_UP(A.cols), BLOCK_ROUND_UP(B.cols), BLOCK_ROUND_UP(B.rows));

}

void Matrix::matMulT1(Matrix const &A, Matrix const &B, Matrix &C) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(C.cols) / dimBlock.x, BLOCK_ROUND_UP(C.rows) / dimBlock.y);

    matMulT1CUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(B.rows), BLOCK_ROUND_UP(B.cols));

}

// __global__ void matSumCUDA(float *A, float *B, float *C, int M, int N) {
    
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {
//         C[y * N + x] = A[y * N + x] + B[y * N + x];
//     }
// }

// void Matrix::matSum(Matrix const &A, Matrix const &B, Matrix &C) {
//     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//     dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

//     matSumCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, A.rows, A.cols);
// }


__global__ void matSubCUDA(float *A, float *B, float *C, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    C[y * N + x] = A[y * N + x] - B[y * N + x];
}

void Matrix::matSub(Matrix const &A, Matrix const &B, Matrix &C) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matSubCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}

__global__ void matElMulCUDA(float *A, float *B, float *C, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    C[y * N + x] = A[y * N + x] * B[y * N + x];
}

void Matrix::matElMul(Matrix const &A, Matrix const &B, Matrix &C) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matElMulCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, C.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}

// __global__ void matScalarMulCUDA(float const e, float *A, float *B, int M, int N) {
    
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {
//         B[y * N + x] = e * A[y * N + x];
//     }
// }

// void Matrix::matScalarMul(float const x, Matrix const &A, Matrix &B) {
//     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//     dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

//     matScalarMulCUDA<<<dimGrid, dimBlock>>>(x, A.buff, B.buff, A.rows, A.cols);
// }

// __global__ void matTCUDA(float *A, float *B, int M, int N) {
    
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {
//         B[x * M + y] = A[y * N + x];
//     }
// }

// void Matrix::matT(Matrix const &A, Matrix &B) {

//     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//     dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

//     matTCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, A.rows, A.cols);
// }

__global__ void matReLUCUDA(float *A, float *B, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    B[y * N + x] = A[y * N + x] > 0 ? A[y * N + x] : 0;
}

void Matrix::matReLU(Matrix const &A, Matrix &B) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matReLUCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}




// __global__ void matSigmoidCUDA(float *A, float *B, int M, int N) {
    
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {
//         B[y * N + x] = 1.0f / (1.0f + expf(-A[y * N + x]));
//     }
// }

// void Matrix::matSigmoid(Matrix const &A, Matrix &B) {
//     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//     dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

//     matSigmoidCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, A.rows, A.cols);
// }

// __global__ void matSigmoidPrimeCUDA(float *A, float *B, int M, int N) {
    
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

//     if (y < M && x < N) {
//         float tmp = 1.0f / (1.0f + expf(-A[y * N + x]));
//         B[y * N + x] = tmp * (1 - tmp);
//     }
// }

// void Matrix::matSigmoidPrime(Matrix const &A, Matrix &B) {
//     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//     dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

//     matSigmoidPrimeCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, A.rows, A.cols);
// }



__global__ void matTanhCUDA(float *A, float *B, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    B[y * N + x] = tanh(A[y * N + x]);
}

void Matrix::matTanh(Matrix const &A, Matrix &B) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matTanhCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}

__global__ void matTanhPrimeCUDA(float *A, float *B, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp = tanh(A[y * N + x]);
    B[y * N + x] = 1 - tmp;
}

void Matrix::matTanhPrime(Matrix const &A, Matrix &B) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matTanhPrimeCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}

__global__ void matReLUPrimeCUDA(float *A, float *B, int M, int N) {
    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    B[y * N + x] = A[y * N + x] >= 0 ? 1 : 0;
}

void Matrix::matReLUPrime(Matrix const &A, Matrix &B) {
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x, BLOCK_ROUND_UP(A.rows) / dimBlock.y);

    matReLUPrimeCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, BLOCK_ROUND_UP(A.rows), BLOCK_ROUND_UP(A.cols));
}

__global__ void matSoftmaxCUDA(float *A, float *B, int M, int N) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < N) {
        float m = A[x];

        for (int j = 0; j < M; ++j) {
            m = fmax(m, A[BLOCK_ROUND_UP(N) * j + x]);
        }

        float sum = 0.0f;
        for (int j = 0; j < M; ++j) {
            sum += expf(A[BLOCK_ROUND_UP(N) * j + x] - m);
        }

        for (int j = 0; j < M; ++j) {
            B[BLOCK_ROUND_UP(N) * j + x] = expf(A[BLOCK_ROUND_UP(N) * j + x] - m) / sum;
        }
    }
}

void Matrix::matColSoftmax(Matrix const &A, Matrix &B) {
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(B.cols) / dimBlock.x);

    matSoftmaxCUDA<<<dimGrid, dimBlock>>>(A.buff, B.buff, A.rows, B.cols);
}


// float Matrix::cost(Matrix const &A, Matrix const &B) {

//     float *tmp_A = new float[A.size()];
//     float *tmp_B = new float[B.size()];

//     hipMemcpy(tmp_A, A.buff, A.size() * sizeof(float), hipMemcpyDeviceToHost);
//     hipMemcpy(tmp_B, B.buff, B.size() * sizeof(float), hipMemcpyDeviceToHost);

//     float sum = 0;
//     for (int i = 0; i < A.cols; ++i) {
//         for (int j = 0; j  < A.rows; ++j) {
//             if (tmp_B[B.cols * j + i]) {
//              sum -= logf(tmp_A[A.cols * j + i]);
//             }
//         }
//     }

//     delete tmp_A;
//     delete tmp_B;

//     return sum / A.cols;
// }

void Matrix::initialize() {

    float *tmp_buff = (float*) calloc(BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols), sizeof(float));
    
    std::mt19937 rng;
    std::uniform_real_distribution<float> distribution(0.0f, 1.0f);

    for(int i = 0; i < this->rows; ++i) {
        for (int j = 0; j < this->cols; ++j) {
            tmp_buff[i * BLOCK_ROUND_UP(this->cols) + j] = distribution(rng);
        }
    }
    hipMemcpy(this->buff, tmp_buff, BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols) * sizeof(float), hipMemcpyHostToDevice);
    free(tmp_buff);
}

void Matrix::initialize(float *buff) {

    if (this->cols == BLOCK_ROUND_UP(this->cols) && this->rows == BLOCK_ROUND_UP(this->rows)) {
        
        hipMemcpy(this->buff, buff, this->rows * this->cols * sizeof(float), hipMemcpyHostToDevice);
        return;
    }

    float *tmp_buff = (float*) calloc(BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols), sizeof(float));

    if (this->cols == BLOCK_ROUND_UP(this->cols)) {
        memcpy(tmp_buff, buff, this->rows * this->cols * sizeof(float));        
    } else {

        for (int i = 0; i < this->rows; ++i) {
            memcpy(tmp_buff + BLOCK_ROUND_UP(this->cols) * i, buff + this->cols * i, this->cols * sizeof(float));
        }
    }

    hipMemcpy(this->buff, tmp_buff, BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols) * sizeof(float), hipMemcpyHostToDevice);
    free(tmp_buff);
}

int Matrix::size() const {
    return this->rows * this->cols;
}
int Matrix::getRows() const {
    return this->rows;
}
int Matrix::getCols() const {
    return this->cols;
}

// Used only for testing
bool Matrix::operator==(Matrix const &other) const {

    if (this->rows != other.rows || this->cols != other.cols) {
        return false;
    }

    unsigned size = BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols);

    float *a = new float[size];
    float *b = new float[size];

    hipMemcpy(a, this->buff, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, other.buff, size * sizeof(float), hipMemcpyDeviceToHost);

    bool equal = true;
    for (int i = 0; i < size; ++i) {
        if (a[i] != b[i]) {
            equal = false;
            break;
        }
    }

    delete a;
    delete b;

    return equal;
}

// Used only for testing
std::string Matrix::toString() const {

    std::stringstream ss;
    ss << *this << std::endl;
    ss << std::fixed << std::setprecision(2);
    
    float *a = new float[BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols)];

    hipMemcpy(a, this->buff, BLOCK_ROUND_UP(this->rows) * BLOCK_ROUND_UP(this->cols) * sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < BLOCK_ROUND_UP(this->rows); ++i) {
        for (int j = 0; j < BLOCK_ROUND_UP(this->cols); ++j) {
            
            if (j) {
                ss << " ";
            }
            
            ss << a[BLOCK_ROUND_UP(this->cols) * i + j];
        }
        ss << std::endl;
    }

    delete a;

    return ss.str();
}

std::ostream& operator<<(std::ostream& stream, Matrix const &matrix) {
    return stream << "Matrix[" << matrix.rows << ", " << matrix.cols << "]";
}