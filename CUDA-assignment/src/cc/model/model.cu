#include "hip/hip_runtime.h"
#include <iostream>
#include "model.hh"

//-----------------------------------------------------------------------------
//                            ACCURACY                                       
//-----------------------------------------------------------------------------

__global__
void accuracyCUDA(float *A, float *B, int *C, int M, int N) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;

    float pred_max = A[x], true_max = B[x];
    int pred_arg_max = 0, true_arg_max = 0;

    for(int j = 0; j < M; ++j) {
        float pred_val = A[j * N + x];
        float true_val = B[j * N + x];
        
        if (pred_val > pred_max) {
            pred_max = pred_val;
            pred_arg_max = j;
        }

        if (true_val > true_max) {
            true_max = true_val;
            true_arg_max = j;
        }
    }

    C[x] = pred_arg_max == true_arg_max;
}

float Model::accuracy(Matrix &pred_vals, Matrix &true_vals) {

    int correct_sum = 0;
    
    int *dev_correct;
    int *correct = new int[pred_vals.getCols()];

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(pred_vals.getCols()) / BLOCK_SIZE);
    
    hipMalloc((void**)&dev_correct, BLOCK_ROUND_UP(pred_vals.getCols()) * sizeof(int));
    
    accuracyCUDA<<<dimGrid, dimBlock>>>
    (pred_vals.buff, true_vals.buff, dev_correct, pred_vals.getRows(), BLOCK_ROUND_UP(pred_vals.getCols()));
    
    hipMemcpy(correct, dev_correct, pred_vals.getCols() * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < pred_vals.getCols(); ++i) {
        correct_sum += correct[i];
    }

    delete correct;
    hipFree(dev_correct);

    return ((float) correct_sum) / (float) pred_vals.getCols();
}

//-----------------------------------------------------------------------------
//                            COST                                       
//-----------------------------------------------------------------------------

__global__
void crossEntropyCostCUDA(float *A, float *B, float *C, int M, int N) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    float cost = 0;

    for (int i = 0; i < M; ++i) {
        if (B[i * N + x] != 0) {
            cost -= logf(A[i * N + x]);
        } 
    }

    C[x] = cost;
}

float Model::crossEntropyCost(Matrix &pred_vals, Matrix &true_vals) {
    
    float cost_sum = 0.0f;

    float *dev_cost;
    float *cost = new float[pred_vals.getCols()];
    
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(pred_vals.getCols()) / BLOCK_SIZE);

    hipMalloc((void**)&dev_cost, BLOCK_ROUND_UP(pred_vals.getCols()) * sizeof(int));

    crossEntropyCostCUDA<<<dimGrid, dimBlock>>>
    (pred_vals.buff, true_vals.buff, dev_cost, pred_vals.getRows(), BLOCK_ROUND_UP(pred_vals.getCols()));

    hipMemcpy(cost, dev_cost, pred_vals.getCols() * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < pred_vals.getCols(); ++i) {
        cost_sum += cost[i];
    }

    delete cost;
    hipFree(dev_cost);

    return cost_sum / (float) pred_vals.getCols();
}

//-----------------------------------------------------------------------------
//                            GET BATCH                                       
//-----------------------------------------------------------------------------

__global__
void getBatchCUDA(float *A, float *B, int AN, int BN, int BNN, int n) {

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < BN) {
        B[y * BNN + x] = A[y * AN + x + n * BN];
    }

}

void getBatch(Matrix &input, Matrix &batch, int n) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(batch.getCols()) / BLOCK_SIZE,  BLOCK_ROUND_UP(batch.getRows()) / BLOCK_SIZE);

    getBatchCUDA<<<dimGrid, dimBlock>>>
    (input.buff, batch.buff, BLOCK_ROUND_UP(input.getCols()), batch.getCols(), BLOCK_ROUND_UP(batch.getCols()),  n);
}

//-----------------------------------------------------------------------------
//                            FIT                                       
//-----------------------------------------------------------------------------

float Model::fit(float *data_x, float *data_y, int len,  int epochs, 
                float learning_rate, float eps, int random) {

    // Initialize input & output
    Matrix data_X(this->input_size, len);
    Matrix data_Y(this->output_size, len);
    data_X.init(data_x);
    data_Y.init(data_y);

    // Initialize layers
    for (Layer *l : this->layers) {
        l->initialize(random);
    }

    // Create timer events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int i;
    float sum_acc = 0;
    
    for (i = 0; i < epochs; ++i) {

        int j;
        float acc = 0, cost = 0;
        hipEventRecord( start, 0 );

        for (j = 0; j < len / this->batch_size; ++j) {

            getBatch(data_X, this->input, j);
            getBatch(data_Y, this->output, j);

            // Forward pass
            Matrix *input = &this->input;
            for (Layer *layer : this->layers) {
                input = &layer->forward_pass(*input);
                
            }

            // Loss Function
            cost += crossEntropyCost(*input, this->output);
            acc += accuracy(*input, this->output);


            // Delta
            Matrix::matSub(*input, this->output, this->delta);

            // Backward pass
            Matrix *delta = &this->delta;
            for (auto it = this->layers.rbegin(); it != this->layers.rend(); ++it) {
                delta = &(*it)->backward_pass(*delta);
            }

            // Update
            for (Layer * layer : this->layers) {
                layer->update(learning_rate);
            }
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);

        cost /= j;
        acc /= j;

        sum_acc += acc;

        std::cout << std::fixed << "epoch " << i + 1 << "/" << epochs << "\t" 
                  << std::setprecision(3) << "time: " << elapsedTime << " ms, "
                  << "cost: " << cost << ", "
                  << std::setprecision(2) << "accuracy: " << acc << std::endl;

        if (cost < eps) {
            ++i;
            break;
        }
    }
        
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return sum_acc / i;
}