#include "hip/hip_runtime.h"
#include <iostream>
#include "model.hh"

//-----------------------------------------------------------------------------
//                            ACCURACY                                       
//-----------------------------------------------------------------------------

__global__
void accuracyCUDA(float *A, float *B, int *C, int M, int N) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;

    float pred_max = A[x], true_max = B[x];
    int pred_arg_max = 0, true_arg_max = 0;

    for(int j = 0; j < M; ++j) {
        float pred_val = A[j * N + x];
        float true_val = B[j * N + x];
        
        if (pred_val > pred_max) {
            pred_max = pred_val;
            pred_arg_max = j;
        }

        if (true_val > true_max) {
            true_max = true_val;
            true_arg_max = j;
        }
    }

    C[x] = pred_arg_max == true_arg_max;
}

float Model::accuracy(Matrix &pred_vals, Matrix &true_vals) {

    int correct_sum = 0;
    
    int *dev_correct;
    int *correct = new int[pred_vals.getCols()];

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(pred_vals.getCols()) / BLOCK_SIZE);
    
    hipMalloc((void**)&dev_correct, BLOCK_ROUND_UP(pred_vals.getCols()) * sizeof(int));
    
    accuracyCUDA<<<dimGrid, dimBlock>>>
    (pred_vals.buff, true_vals.buff, dev_correct, pred_vals.getRows(), BLOCK_ROUND_UP(pred_vals.getCols()));
    
    hipMemcpy(correct, dev_correct, pred_vals.getCols() * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < pred_vals.getCols(); ++i) {
        correct_sum += correct[i];
    }

    delete correct;
    hipFree(dev_correct);

    return ((float) correct_sum) / (float) pred_vals.getCols();
}

//-----------------------------------------------------------------------------
//                            COST                                       
//-----------------------------------------------------------------------------

__global__
void crossEntropyCostCUDA(float *A, float *B, float *C, int M, int N) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    float cost = 0;

    for (int i = 0; i < M; ++i) {
        if (B[i * N + x] != 0) {
            cost -= logf(A[i * N + x]);
        } 
    }

    C[x] = cost;
}

float Model::crossEntropyCost(Matrix &pred_vals, Matrix &true_vals) {
    
    float cost_sum = 0.0f;

    float *dev_cost;
    float *cost = new float[pred_vals.getCols()];
    
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(BLOCK_ROUND_UP(pred_vals.getCols()) / BLOCK_SIZE);

    hipMalloc((void**)&dev_cost, BLOCK_ROUND_UP(pred_vals.getCols()) * sizeof(int));

    crossEntropyCostCUDA<<<dimGrid, dimBlock>>>
    (pred_vals.buff, true_vals.buff, dev_cost, pred_vals.getRows(), BLOCK_ROUND_UP(pred_vals.getCols()));

    hipMemcpy(cost, dev_cost, pred_vals.getCols() * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < pred_vals.getCols(); ++i) {
        cost_sum += cost[i];
    }

    delete cost;
    hipFree(dev_cost);

    return cost_sum / (float) pred_vals.getCols();
}

//-----------------------------------------------------------------------------
//                            FIT                                       
//-----------------------------------------------------------------------------

void Model::fit(float *data_x, float *data_y, int len,  int epochs, 
                float learning_rate, float eps, int random) {

    // Initialize input & output
    this->input.init(data_x);
    this->output.init(data_y);

    // Initialize layers
    for (Layer *l : this->layers) {
        l->initialize(1);
    }

    // Create timer events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    for (int i = 0; i < epochs; ++i) {
        
        hipEventRecord( start, 0 );

        // Forward pass
        Matrix *input = &this->input;
        for (Layer *layer : this->layers) {
            input = &layer->forward_pass(*input);
            
        }
        // std::cout << "OUTPUT VALUES" << std::endl;
        // std::cout << input->toString() << std::endl;
        // std::cout << "---------------------" << std::endl;

        // std::cout << "CORRECT VALUES" << std::endl;
        // std::cout << this->output.toString() << std::endl;
        // std::cout << "---------------------" << std::endl;

        // hipDeviceSynchronize();

        // Loss Function
        float tmp_cost = crossEntropyCost(*input, this->output);
        float acc = accuracy(*input, this->output);

        // std::cout << "OUTPUT VALUES" << std::endl;
        // std::cout << input->toString() << std::endl;
        // std::cout << "---------------------" << std::endl;


        // Delta
        Matrix::matSub(*input, this->output, this->delta);

        // Backward pass
        Matrix *delta = &this->delta;
        for (auto it = this->layers.rbegin(); it != this->layers.rend(); ++it) {
            // std::cout << "DELTA" << std::endl;
            // std::cout << delta->toString() << std::endl;
            // std::cout << "---------------------" << std::endl;
            delta = &(*it)->backward_pass(*delta);
        }

        // Update
        for (Layer * layer : this->layers) {
            layer->update(learning_rate);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        std::cout << std::fixed << "epoch " << i + 1 << "/" << epochs << "\t" 
                  << std::setprecision(3) << "time: " << elapsedTime << " ms, "
                  << "cost: " << tmp_cost << ", "
                  << std::setprecision(2) << "accuracy: " << acc << std::endl;
    }
        
    hipEventDestroy(start);
    hipEventDestroy(stop);
}